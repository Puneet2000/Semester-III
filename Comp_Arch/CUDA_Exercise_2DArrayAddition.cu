#define N 512
#define NUM_BLOCKS 16
#define NUM_THREADS 48
//Do not change above three lines.

//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated. 

/*Remember the following guidelines to avoid losing marks
This exercise is quite simple. The only tricky part is that total number of threads (NUM_BLOCKS*NUM_THREADS) may be different (higher or lower) from N.

Index of an array should not exceed the array size. 

No output array-element should be computed more than once

No marks will be given if the program does not compile or run (TAs will not debug your program at all)

Do not change the name of any variable that we have introduced. 
*/



#include <hip/hip_runtime.h>
#include <stdio.h>

//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than array-element or not process any array-element at all. 
__global__ void MatrixAddition(int *A, int *B , int *C )
{
	int op_per_thread = (N*N)/(NUM_BLOCKS*NUM_THREADS);
	if((N*N)%(NUM_BLOCKS*NUM_THREADS)!=0)
		op_per_thread +=1;

	int thread_index = threadIdx.x + blockIdx.x*blockDim.x;
  printf("%d",blockDim.x);
	for(int i=0;i<op_per_thread;i++){
		int index = thread_index*op_per_thread + i;
		if(index < N*N)
			*(C+index) = *(B+index) + *(A+index);
	}
}

int main (int argc, char **argv) {
  int A[N][N], B[N][N], C[N][N];
  int *d_A, *d_B, *d_C; // These are the copies of A, B and C on the GPU
  int *h_C;       // This is a host copy of the output of B from the GPU
  int i, j;
  h_C = (int *)malloc(N*N*sizeof(int));
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      A[i][j] = i+j;
      B[i][j]= 2*j-1;      
    }
  }

  // sequential implementation of main computation
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      C[i][j] = A[i][j]+B[i][j];
    }
  }

  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A, d_B and d_C
  hipMalloc((void **)&d_A, N*N*sizeof(int));
  hipMalloc((void **)&d_B, N*N*sizeof(int));
  hipMalloc((void **)&d_C, N*N*sizeof(int));
  // TODO: COPY A TO d_A
  // TODO: COPY B TO d_B
  hipMemcpy(d_A,&A[0], N*N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_B,&B[0], N*N*sizeof(int), hipMemcpyHostToDevice);


  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
   //Use NUM_BLOCKS blocks, each with NUM_THREADS  threads
  MatrixAddition<<<NUM_BLOCKS,NUM_THREADS>>>(d_A,d_B,d_C);

  // TODO: COPY d_C BACK FROM GPU to CPU in variable h_C

  hipMemcpy(h_C,d_C, N*N*sizeof(int), hipMemcpyDeviceToHost);

  // TODO: Verify result is correct by comparing
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
    //TODO: compare each element of h_C and C by subtracting them
        //print only those elements for which the above subtraction is non-zero
    int diff = C[i][j] - *(h_C + N*i +j);
    if(diff!=0)
    	printf("error at %d and %d\n",C[i][j],*(h_C + N*i +j));
    }
   }
    //IF even one element of h_C and C differ, report an error.
    //Otherwise, there is no error.
    //If your program is correct, no error should occur.
   free(h_C);
   hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);
}

